#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>


#define NO_OF_THREADS_IN_BLOCK 1024
#define NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES NO_OF_THREADS_IN_BLOCK*2
#define NO_OF_BLOCKS 2048
#define N NO_OF_BLOCKS*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES

__global__ void prefixSum(float *g_odata, float *g_idata, float* sum, int n) { 
    __shared__ float temp[NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES];  // allocated on invocation 

    int thid = threadIdx.x; 
    int offset = 1;

    temp[2*thid] = g_idata[blockIdx.x*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES+2*thid]; // load input into shared memory 
    temp[2*thid+1] = g_idata[blockIdx.x*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES+2*thid+1];

    for (int d = n>>1; d > 0; d >>= 1)                    
    // build sum in place up the tree 
    { 
        __syncthreads();    
        
        if (thid < d){ 

            int ai = offset*(2*thid+1)-1;     
            int bi = offset*(2*thid+2)-1;  

            temp[bi] += temp[ai];        
        }    
        
        offset *= 2; 
    }


    if (thid == 0) { // clear the last element
        sum[blockIdx.x] = temp[n-1];
        temp[n - 1] = 0; 
    } 


    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan 
    {      
        offset >>= 1;      
        __syncthreads();     
    
        if (thid < d){ 
            int ai = offset*(2*thid+1)-1;     
            int bi = offset*(2*thid+2)-1; 
        
            float t = temp[ai]; 
            temp[ai] = temp[bi]; 
            temp[bi] += t;       
        } 
    } 

    __syncthreads();

    g_odata[blockIdx.x*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES+2*thid] = temp[2*thid]; // write results to device memory      
    g_odata[blockIdx.x*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES+2*thid+1] = temp[2*thid+1];
}

//add sum[block_index] to g_odata
__global__ void addKernel(float *g_odata, float *sum){
    if(blockIdx.x > 0){
        g_odata[blockIdx.x*NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES+threadIdx.x] += sum[blockIdx.x];
    }
}

int main(){
    float *g_odata, *g_idata, *sum, *cumSum,*var;

    hipMallocManaged(&g_odata, N*sizeof(float));
    hipMallocManaged(&g_idata, N*sizeof(float));
    hipMallocManaged(&sum, NO_OF_BLOCKS*sizeof(float));
    hipMallocManaged(&cumSum, NO_OF_BLOCKS*sizeof(float));
    hipMallocManaged(&var, sizeof(float));
    
    cumSum[0] = 0;

    for(int i = 0; i < N; i++)
        g_idata[i] = i;

    hipMemPrefetchAsync(g_idata, N*sizeof(float), 0);      
    hipMemPrefetchAsync(g_odata, N*sizeof(float), 0);       
    hipMemPrefetchAsync(sum, NO_OF_BLOCKS*sizeof(float), 0);       
    hipMemPrefetchAsync(cumSum, NO_OF_BLOCKS*sizeof(float), 0);        
    hipMemPrefetchAsync(var, sizeof(float), 0);

    auto start = std::chrono::system_clock::now();

    prefixSum<<<NO_OF_BLOCKS, NO_OF_THREADS_IN_BLOCK>>>(g_odata, g_idata, sum, NO_OF_ELEMENTS_EACH_BLOCK_PROCESSES);

    hipDeviceSynchronize();

    if(NO_OF_BLOCKS > 1){
        prefixSum<<<1, NO_OF_THREADS_IN_BLOCK>>>(cumSum, sum, var, NO_OF_BLOCKS);
        hipDeviceSynchronize();

        addKernel<<<NO_OF_BLOCKS, NO_OF_THREADS_IN_BLOCK*2>>>(g_odata, cumSum);
        hipDeviceSynchronize();        
    }

    auto end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "Total Number of Elements (N): " << N << "\n Number of Blocks: " << NO_OF_BLOCKS << "\n Number of Threads in Each Block: " << NO_OF_THREADS_IN_BLOCK << "\n Time: " << elapsed.count() << std::endl;

    hipFree(g_odata);
    hipFree(g_idata);
    hipFree(sum);
    hipFree(cumSum);
    hipFree(var);

    return 0;
}
